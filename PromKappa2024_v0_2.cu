#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>
#include <iostream>
#include <thread>
#include <vector>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sstream>
#include <fstream>

#define CYCLE_SIZE 4096

struct ChunkContext {
    std::string promoter_name = "unnamed";
    std::string promoter = "";
    int len;
    int window_size = 30;
    int id_cycle = 0;
    bool export_kappaic = false;
    FILE* export_kappaic_file;
    bool export_kappaic_medium = false;
    FILE* export_kappaic_medium_file;
};

struct Flags {
    std::string output_file = "";
    std::string input_file = "";
    std::string dna = "";
    int window_size = 30;
    bool create_kappa_ic = false;
    bool record_medium = false;
    int len_dna = -1;
    std::string name = "";
};

__global__ void moveWindow(const char* input, float** output, int n, int window_size, int id_cycle) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float m = 0;
    float t = 0;

    if (tid <= n) {
        output[0][tid] = tid + id_cycle*(CYCLE_SIZE-29);
        output[1][tid] = tid + id_cycle*(CYCLE_SIZE-29);
    
        float counter = 0;

        for (int u = 1; u < window_size; u++) {
            for (int i = 0; i < window_size - u; i++) {
                if (input[tid + i] == input[tid + i + u]) {
                    m += 1;
                }
            }

            t += (m / double(window_size - u)) * 100.0;
            m = 0;
            
            if (input[tid + u - 1] == 'G' || input[tid + u - 1] == 'C') {
                counter += 1;
            }
        }

        output[0][tid] = counter/float(window_size);
        output[1][tid] = (100.0 - (t / float(window_size - 1)));
    }
}

void createPatternForChunk(ChunkContext context) {
    int id;
    hipGetDevice(&id);

    //std::cout << context.promoter.length() << std::endl;

    char *a = NULL; // for inputs
    float **b = NULL; // for outputs
    
    hipMallocManaged(&a, context.len * sizeof(char));
    hipMallocManaged(&b, 2 * sizeof(float*));

    for (int i = 0; i < 2; ++i) {
        hipMallocManaged(&b[i], context.len * sizeof(float));
    }

    hipMemcpy(a, context.promoter.c_str(), context.len * sizeof(char), hipMemcpyHostToDevice);

    int BLOCK_SIZE = 256;
    int GRID_SIZE = (context.len + BLOCK_SIZE - 1) / BLOCK_SIZE;

    moveWindow<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, context.len, context.window_size, context.id_cycle);

    hipDeviceSynchronize();

    if (context.export_kappaic) {
        for(int i = 0; i < context.len-context.window_size; i++) {
            fprintf(context.export_kappaic_file, "%f, %f\n", b[0][i], b[1][i]);
        }
    }
    if (context.export_kappaic_medium) {
        float gpc = 0;
        float ic = 0;
        for(int i = 0; i < context.len-context.window_size; i++) {
            gpc += b[0][i];
            ic += b[1][i];
        }
        
        fprintf(
            context.export_kappaic_medium_file, 
            "%s, %f, %f\n",  
            context.promoter_name.c_str(),
            gpc/(context.len - context.window_size), 
            ic/(context.len - context.window_size)
        );
    }

    hipFree(a);
    for (int i = 0; i < 2; ++i) {
        hipFree(b[i]);
    }
    hipFree(b);
    return;
}

bool directoryExists(std::string directoryName) {
    struct stat info;
    if (stat(directoryName.c_str(), &info) != 0) {
        return false;
    }
    return (info.st_mode & S_IFDIR) != 0;
}

FILE* createFile(std::string folder_name, std::string file_name) {
    if (!directoryExists(folder_name)) {
        if (mkdir(folder_name.c_str(), 0777) == -1) {
            perror("Error creating directory");
            NULL;
        }
    } 
    
    int length = std::snprintf(nullptr, 0, "%s/%s", folder_name.c_str(), file_name.c_str());
    char buffer[length + 1];
    std::sprintf(buffer, "%s/%s", folder_name.c_str(), file_name.c_str());

    // Create and open the text file in write mode
    FILE *file = fopen(buffer, "w");
    if (file == NULL) {
        perror("Error creating file");
        NULL;
    }
    return file;
}

std::vector<std::string> splitString(const char* input) {
    std::istringstream iss(input);
    std::vector<std::string> tokens;
    std::string token;
    while (iss >> token) {
        tokens.push_back(token);
    }
    return tokens;
}

void StartPattern(Flags thisFlag) {
    if (thisFlag.dna != "") {
        struct ChunkContext newChunkContext;
        newChunkContext.promoter = thisFlag.dna;
        newChunkContext.window_size = thisFlag.window_size;
        newChunkContext.export_kappaic = thisFlag.create_kappa_ic;
        newChunkContext.export_kappaic_medium = thisFlag.record_medium;
        newChunkContext.len = thisFlag.len_dna;
        if (thisFlag.name != "") {
            newChunkContext.promoter_name = thisFlag.name;
        }
        char* kappa_ic_file_name = (char*)malloc(strlen(newChunkContext.promoter_name.c_str()));
        strcpy(kappa_ic_file_name, newChunkContext.promoter_name.c_str());
        char kappa_ic_medium_file_name[] = "kappa_ic_medium";
        FILE* ic_file = createFile(thisFlag.output_file, kappa_ic_file_name);
        FILE* medium_file = createFile(thisFlag.output_file, kappa_ic_medium_file_name);

        if (ic_file && medium_file) {
            newChunkContext.export_kappaic_file = ic_file;
            newChunkContext.export_kappaic_medium_file = medium_file;

            createPatternForChunk(newChunkContext);

            fclose(ic_file);
            fclose(medium_file);
        } else {
            printf("Error: Failed to create output files.\n");
        }
    } else {
        std::string dna;
        std::vector<std::string> token;
        FILE* datafile = fopen(thisFlag.input_file.c_str(), "r");
        FILE* ic_file;
        FILE* medium_file;

        if (thisFlag.record_medium) {
            char kappa_ic_medium_file_name[] = "kappa_ic_medium";
            medium_file = createFile(thisFlag.output_file, kappa_ic_medium_file_name);
        }

        if (datafile != NULL) {
            char line[1000]; // Assuming a maximum line length of 1000 characters
            while (fgets(line, sizeof(line), datafile)) {
                if (line[0] == '>') {
                    if (token.size() != 0) {
                        struct ChunkContext newChunkContext;
                        newChunkContext.promoter = dna;
                        newChunkContext.window_size = thisFlag.window_size;
                        newChunkContext.export_kappaic = thisFlag.create_kappa_ic;
                        newChunkContext.export_kappaic_medium = thisFlag.record_medium;
                        newChunkContext.len = dna.length();
                        newChunkContext.promoter_name = token.at(1);
                        char* kappa_ic_file_name = (char*)malloc(strlen(newChunkContext.promoter_name.c_str()));
                        strcpy(kappa_ic_file_name, newChunkContext.promoter_name.c_str());
                        ic_file = createFile(thisFlag.output_file, kappa_ic_file_name);                    
                        
                        newChunkContext.export_kappaic_file = ic_file;
                        newChunkContext.export_kappaic_medium_file = medium_file;
                        
                        createPatternForChunk(newChunkContext);
                        
                        fclose(ic_file);
                    }
                    dna = "";
                    token = splitString(line);
                    //std::cout << token.at(1) << std::endl;
                } else {
                    dna += std::string(line);
                    if (dna.at(dna.size()-1) == '\n') {
                        dna = dna.substr(0, dna.size() - 1);
                    }
                }
            }
            struct ChunkContext newChunkContext;
            newChunkContext.promoter = dna;
            newChunkContext.window_size = thisFlag.window_size;
            newChunkContext.export_kappaic = thisFlag.create_kappa_ic;
            newChunkContext.export_kappaic_medium = thisFlag.record_medium;
            newChunkContext.len = dna.length();
            newChunkContext.promoter_name = token.at(1);
            char* kappa_ic_file_name = (char*)malloc(strlen(newChunkContext.promoter_name.c_str()));
            strcpy(kappa_ic_file_name, newChunkContext.promoter_name.c_str());
            ic_file = createFile(thisFlag.output_file, kappa_ic_file_name);
        
            newChunkContext.export_kappaic_file = ic_file;
            newChunkContext.export_kappaic_medium_file = medium_file;
            
            createPatternForChunk(newChunkContext);
        
            fclose(ic_file);            
            fclose(medium_file);
            fclose(datafile);
        } else {
            fprintf(stderr, "Unable to open file: %s\n", thisFlag.input_file.c_str());
        }   
    }
    return;
}

int main(int argc, char* argv[]) {
    struct Flags this_Flag;
    bool helpRequested = false;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-o") == 0 && i + 1 < argc) {
            this_Flag.output_file = argv[i + 1];
        } else if (strcmp(argv[i], "-i") == 0  && i + 1 < argc) {
            this_Flag.input_file = argv[i + 1];
        } else if (strcmp(argv[i], "-dna") == 0 && i + 1 < argc) {
            this_Flag.dna = argv[i + 1];
            this_Flag.len_dna = this_Flag.dna.length();
            i++;
        } else if (strcmp(argv[i], "-ws") == 0 && i + 1 < argc) {
            this_Flag.window_size = atoi(argv[i + 1]);
            i++;
        } else if (strcmp(argv[i], "-name") == 0 && i + 1 < argc) {
            this_Flag.name = argv[i + 1];
            i++;
        } else if (strcmp(argv[i], "-h") == 0) {
            helpRequested = true;
        } else if (strcmp(argv[i], "-m") == 0) {
            this_Flag.record_medium = true;
        } else if (strcmp(argv[i], "-ic") == 0) {
            this_Flag.create_kappa_ic = true;
        }

    }

    // Print help message if requested
    if (helpRequested) {
        printf("Options:\n");
        printf("  -i <file>            : Input file\n");
        printf("  -dna <dna string>    : Dna String as argument\n");
        printf("  -h                   : Display this help message\n");
        printf("  -o <file>            : Specify output folder\n");
        printf("  -m                   : Process for medium value\n");
        printf("  -ic                  : Save KappaIC\n");
        printf("  -name <str>          : name of promoter\n");
        printf("  -ws <int>            : window size\n");
        return 0;
    }

    StartPattern(this_Flag);

    return 0;
}